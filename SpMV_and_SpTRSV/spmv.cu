#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <iostream>
#include <vector>
#include <chrono>
#include <fstream>
#include <sstream>
#include <cstdlib>
#include <ctime>
#include <cmath>

// Define warp_reduce function
template <typename data_type>
__device__ data_type warp_reduce(data_type val) {
    for (int offset = 16; offset > 0; offset /= 2)
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    return val;
}

template <typename data_type>
__global__ void csr_spmv_vector_kernel(
    unsigned int n_rows,
    const unsigned int* col_ids,
    const unsigned int* row_ptr,
    const data_type* data,
    const data_type* x,
    data_type* y)
{
    const unsigned int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int warp_id = thread_id / 32;
    const unsigned int lane = thread_id % 32;

    const unsigned int row = warp_id; ///< One warp per row

    data_type dot = 0;
    if (row < n_rows) {
        const unsigned int row_start = row_ptr[row];
        const unsigned int row_end = row_ptr[row + 1];

        for (unsigned int element = row_start + lane; element < row_end; element += 32)
            dot += data[element] * x[col_ids[element]];
    }

    dot = warp_reduce(dot);

    if (lane == 0 && row < n_rows) {
        y[row] = dot;
    }
}

// Host function to measure performance
template <typename data_type>
void csr_spmv_vector_host(
    unsigned int n_rows,
    const std::vector<unsigned int>& h_col_ids,
    const std::vector<unsigned int>& h_row_ptr,
    const std::vector<data_type>& h_data,
    const std::vector<data_type>& h_x,
    std::vector<data_type>& h_y,
    float& execution_time, float& throughput, float& bandwidth)
{
    // Allocate device memory
    unsigned int* d_col_ids;
    unsigned int* d_row_ptr;
    data_type* d_data;
    data_type* d_x;
    data_type* d_y;

    hipMalloc(&d_col_ids, h_col_ids.size() * sizeof(unsigned int));
    hipMalloc(&d_row_ptr, h_row_ptr.size() * sizeof(unsigned int));
    hipMalloc(&d_data, h_data.size() * sizeof(data_type));
    hipMalloc(&d_x, h_x.size() * sizeof(data_type));
    hipMalloc(&d_y, h_y.size() * sizeof(data_type));

    // Copy data from host to device
    hipMemcpy(d_col_ids, h_col_ids.data(), h_col_ids.size() * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_row_ptr, h_row_ptr.data(), h_row_ptr.size() * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_data, h_data.data(), h_data.size() * sizeof(data_type), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x.data(), h_x.size() * sizeof(data_type), hipMemcpyHostToDevice);

    // Define kernel launch parameters
    const unsigned int threads_per_block = 256; // Number of threads per block
    const unsigned int warps_per_block = threads_per_block / 32; // Number of warps per block
    const unsigned int num_blocks = (n_rows + warps_per_block - 1) / warps_per_block;

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Launch the kernel multiple times for averaging
    const int num_experiments = 5;
    hipEventRecord(start);
    for (int i = 0; i < num_experiments; ++i) {
        csr_spmv_vector_kernel<data_type> << <num_blocks, threads_per_block >> > (
            n_rows, d_col_ids, d_row_ptr, d_data, d_x, d_y);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Measure elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    milliseconds /= num_experiments; // Average time per kernel launch
    execution_time = milliseconds;

    // Copy results back to host
    hipMemcpy(h_y.data(), d_y, h_y.size() * sizeof(data_type), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_col_ids);
    hipFree(d_row_ptr);
    hipFree(d_data);
    hipFree(d_x);
    hipFree(d_y);

    // Calculate performance metrics
    unsigned int nnz = h_data.size(); // Number of non-zero elements
    throughput = 2.0f * nnz / (milliseconds / 1000.0f); // FLOPS (2 operations per non-zero element)
    float memory_bytes = (h_col_ids.size() + h_row_ptr.size() + h_data.size() + h_x.size() + h_y.size()) * sizeof(data_type);
    bandwidth = memory_bytes / (milliseconds / 1000.0f) / (1024.0f * 1024.0f * 1024.0f); // GB/s
}

// cuSPARSE implementation
void cusparse_spmv(
    unsigned int n_rows,
    unsigned int n_cols,
    unsigned int nnz,
    const std::vector<unsigned int>& h_row_ptr,
    const std::vector<unsigned int>& h_col_ids,
    const std::vector<float>& h_data,
    const std::vector<float>& h_x,
    std::vector<float>& h_y,
    float& execution_time, float& throughput, float& bandwidth)
{
    // Allocate device memory
    unsigned int* d_row_ptr;
    unsigned int* d_col_ids;
    float* d_data;
    float* d_x;
    float* d_y;

    hipMalloc(&d_row_ptr, h_row_ptr.size() * sizeof(unsigned int));
    hipMalloc(&d_col_ids, h_col_ids.size() * sizeof(unsigned int));
    hipMalloc(&d_data, h_data.size() * sizeof(float));
    hipMalloc(&d_x, h_x.size() * sizeof(float));
    hipMalloc(&d_y, h_y.size() * sizeof(float));

    hipMemcpy(d_row_ptr, h_row_ptr.data(), h_row_ptr.size() * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_ids, h_col_ids.data(), h_col_ids.size() * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_data, h_data.data(), h_data.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x.data(), h_x.size() * sizeof(float), hipMemcpyHostToDevice);

    // cuSPARSE handle
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    // Matrix descriptor
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    hipsparseCreateCsr(&matA, n_rows, n_cols, nnz, d_row_ptr, d_col_ids, d_data,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    hipsparseCreateDnVec(&vecX, n_cols, d_x, HIP_R_32F);
    hipsparseCreateDnVec(&vecY, n_rows, d_y, HIP_R_32F);

    // Buffer size and allocation
    size_t bufferSize;
    void* dBuffer;
    float alpha = 1.0;
    float beta = 0.0;
    hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY,
        HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    // Timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int num_experiments = 5;
    hipEventRecord(start);
    for (int i = 0; i < num_experiments; ++i) {
        hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY,
            HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    milliseconds /= num_experiments; // Average time per kernel launch
    execution_time = milliseconds;

    // Copy result back to host
    hipMemcpy(h_y.data(), d_y, h_y.size() * sizeof(float), hipMemcpyDeviceToHost);

    // Calculate performance metrics
    throughput = 2.0f * nnz / (milliseconds / 1000.0f); // FLOPS
    float memory_bytes = (h_row_ptr.size() + h_col_ids.size() + h_data.size() + h_x.size() + h_y.size()) * sizeof(float);
    bandwidth = memory_bytes / (milliseconds / 1000.0f) / (1024.0f * 1024.0f * 1024.0f); // GB/s

    // Cleanup
    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnVec(vecX);
    hipsparseDestroyDnVec(vecY);
    hipsparseDestroy(handle);
    hipFree(dBuffer);
    hipFree(d_row_ptr);
    hipFree(d_col_ids);
    hipFree(d_data);
    hipFree(d_x);
    hipFree(d_y);
}

void read_matrix_market(const std::string& filename,
    std::vector<unsigned int>& row_ptr,
    std::vector<unsigned int>& col_ids,
    std::vector<float>& data,
    unsigned int& n_rows,
    unsigned int& n_cols) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Error: Could not open file " << filename << std::endl;
        return;
    }

    std::string line;
    // Skip comments
    while (std::getline(file, line)) {
        if (line[0] != '%') break;
    }

    // Read matrix dimensions
    std::istringstream iss(line);
    unsigned int nnz;
    iss >> n_rows >> n_cols >> nnz;

    std::vector<unsigned int> row_indices(nnz);
    std::vector<unsigned int> col_indices(nnz);
    std::vector<float> values(nnz);

    // Read COO data
    for (unsigned int i = 0; i < nnz; ++i) {
        file >> row_indices[i] >> col_indices[i] >> values[i];
        row_indices[i]--; // Convert to 0-based indexing
        col_indices[i]--; // Convert to 0-based indexing
    }

    // Convert COO to CSR
    row_ptr.resize(n_rows + 1, 0);
    col_ids.resize(nnz);
    data.resize(nnz);

    for (unsigned int i = 0; i < nnz; ++i) {
        row_ptr[row_indices[i] + 1]++;
    }

    for (unsigned int i = 1; i <= n_rows; ++i) {
        row_ptr[i] += row_ptr[i - 1];
    }

    std::vector<unsigned int> temp_row_ptr = row_ptr;
    for (unsigned int i = 0; i < nnz; ++i) {
        unsigned int row = row_indices[i];
        unsigned int dest = temp_row_ptr[row]++;
        col_ids[dest] = col_indices[i];
        data[dest] = values[i];
    }
}

template <typename data_type>
bool compare_results(unsigned int y_size, const data_type* a, const data_type* b) {
    data_type numerator = 0.0;
    data_type denominator = 0.0;

    for (unsigned int i = 0; i < y_size; i++) {
        numerator += (a[i] - b[i]) * (a[i] - b[i]);
        denominator += b[i] * b[i];
    }

    const data_type error = numerator / denominator;

    if (error > 1e-9) {
        std::cerr << "ERROR: Mean Squared Error = " << error << std::endl;

        for (unsigned int i = 0; i < y_size; i++) {
            if (std::abs(a[i] - b[i]) > 1e-8) {
                std::cerr << "Mismatch at index " << i << ": a[" << i << "] = " << a[i] << ", b[" << i << "] = " << b[i] << std::endl;
                break; // Print the first mismatch and exit
            }
        }

        std::cerr.flush();
        return false; // Results do not match
    }

    std::cout << "Results match! Mean Squared Error = " << error << std::endl;
    return true; // Results match
}


int main() {
    std::string filename = "matrices/ML_Laplace.mtx"; // NEED TO MANUALLY CHANGE PATH TO RUN THE CODE
    std::vector<unsigned int> row_ptr, col_ids;
    std::vector<float> data;
    unsigned int n_rows, n_cols;

    // Read the matrix from the Matrix Market file and convert it to CSR format
    read_matrix_market(filename, row_ptr, col_ids, data, n_rows, n_cols);

    // Generate a randomized input vector h_x
    std::vector<float> h_x(n_cols);
    std::vector<float> h_y_manual(n_rows, 0.0f); // Result from manual implementation
    std::vector<float> h_y_cusparse(n_rows, 0.0f); // Result from cuSPARSE implementation
    srand(static_cast<unsigned int>(time(NULL))); // Seed the random number generator
    for (unsigned int i = 0; i < n_cols; ++i) {
        h_x[i] = static_cast<float>(rand()) / RAND_MAX; // Random value between 0 and 1
    }

    // Performance metrics
    float manual_time, manual_throughput, manual_bandwidth;
    float cusparse_time, cusparse_throughput, cusparse_bandwidth;

    // Call the manual CSR SpMV host function
    csr_spmv_vector_host<float>(n_rows, col_ids, row_ptr, data, h_x, h_y_manual, manual_time, manual_throughput, manual_bandwidth);

    // Call the cuSPARSE SpMV function
    cusparse_spmv(n_rows, n_cols, data.size(), row_ptr, col_ids, data, h_x, h_y_cusparse, cusparse_time, cusparse_throughput, cusparse_bandwidth);

    bool results_match = compare_results(n_rows, h_y_manual.data(), h_y_cusparse.data());


    if (results_match) {
        std::cout << "Results match!" << std::endl;
        std::cout << "Manual Implementation Performance:" << std::endl;
        std::cout << "Execution time (ms): " << manual_time << std::endl;
        std::cout << "Throughput (GFLOPS): " << manual_throughput / 1e9 << std::endl;
        std::cout << "Memory bandwidth (GB/s): " << manual_bandwidth << std::endl;

        std::cout << "cuSPARSE Implementation Performance:" << std::endl;
        std::cout << "Execution time (ms): " << cusparse_time << std::endl;
        std::cout << "Throughput (GFLOPS): " << cusparse_throughput / 1e9 << std::endl;
        std::cout << "Memory bandwidth (GB/s): " << cusparse_bandwidth << std::endl;
    }

    return 0;
}

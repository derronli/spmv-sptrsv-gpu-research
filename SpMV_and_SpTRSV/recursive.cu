#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <iostream>
#include <vector>
#include <chrono>
#include <fstream>
#include <sstream>
#include <cstdlib>
#include <ctime>
#include <cmath>
#include <tuple>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <>


// Define a sparse matrix in CSR format
struct CSRMatrix {
    int rows;                // Number of rows
    int cols;                // Number of columns
    int nnz;                 // Number of non-zero elements
    int* d_rowPtr;           // Row pointers (device memory)
    int* d_colIdx;           // Column indices (device memory)
    float* d_values;         // Non-zero values (device memory)
};

// Function prototypes
void spTRSVRecursiveBlock(const CSRMatrix& tri, float* d_b, float* d_x, int depth);
CSRMatrix extractSubMatrix(const CSRMatrix& mat, int startRow, int endRow, int startCol, int endCol);
void allocateCSRMatrix(CSRMatrix& mat, int rows, int cols, int nnz);
void freeCSRMatrix(CSRMatrix& mat);
__global__ void spTRSVKernel(const int* rowPtr, const int* colIdx, const float* values, const float* b, float* x, int rows);
__global__ void spMVKernel(const int* rowPtr, const int* colIdx, const float* values, const float* x, float* b, int rows);

// Host function to measure performance for recursive block SpTRSV
void sptrsv_recursive_block_host(
    unsigned int n_rows,
    const std::vector<unsigned int>& h_row_ptr,
    const std::vector<unsigned int>& h_col_ids,
    const std::vector<float>& h_data,
    const std::vector<float>& h_b,
    std::vector<float>& h_x,
    float& execution_time, float& throughput, float& bandwidth)
{
    // Allocate device memory for the matrix
    CSRMatrix tri;
    allocateCSRMatrix(tri, n_rows, n_rows, h_data.size());

    // Copy data to device
    hipMemcpy(tri.d_rowPtr, h_row_ptr.data(), (n_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(tri.d_colIdx, h_col_ids.data(), h_col_ids.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(tri.d_values, h_data.data(), h_data.size() * sizeof(float), hipMemcpyHostToDevice);

    // Allocate device memory for vectors
    float* d_b, * d_x;
    hipMalloc((void**)&d_b, n_rows * sizeof(float));
    hipMalloc((void**)&d_x, n_rows * sizeof(float));
    hipMemcpy(d_b, h_b.data(), n_rows * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_x, 0, n_rows * sizeof(float));

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Recursion depth (hardcoded to 2)
    int depth = 2;

    // Launch the recursive block SpTRSV
    const int num_experiments = 5;
    hipEventRecord(start);
    for (int i = 0; i < num_experiments; ++i) {
        spTRSVRecursiveBlock(tri, d_b, d_x, depth);
        hipDeviceSynchronize();
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Measure elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    milliseconds /= num_experiments; // Average time per kernel launch
    execution_time = milliseconds;

    // Copy result back to host
    hipMemcpy(h_x.data(), d_x, n_rows * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    freeCSRMatrix(tri);
    hipFree(d_b);
    hipFree(d_x);

    // Calculate performance metrics
    unsigned int nnz = h_data.size(); // Number of non-zero elements
    throughput = 2.0f * nnz / (milliseconds / 1000.0f); // FLOPS (2 operations per non-zero element)
    float memory_bytes = (h_row_ptr.size() + h_col_ids.size() + h_data.size() + h_b.size() + h_x.size()) * sizeof(float);
    bandwidth = memory_bytes / (milliseconds / 1000.0f) / (1024.0f * 1024.0f * 1024.0f); // GB/s
}

// CUDA kernel for sparse triangular solve (SpTRSV)
__global__ void spTRSVKernel(const int* rowPtr, const int* colIdx, const float* values, const float* b, float* x, int rows) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        float sum = b[row];
        for (int j = rowPtr[row]; j < rowPtr[row + 1] - 1; ++j) {
            sum -= values[j] * x[colIdx[j]];
        }
        x[row] = sum / values[rowPtr[row + 1] - 1]; // Diagonal element
    }
}

// CUDA kernel for sparse matrix-vector multiplication (SpMV)
__global__ void spMVKernel(const int* rowPtr, const int* colIdx, const float* values, const float* x, float* b, int rows) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        float sum = 0.0f;
        for (int j = rowPtr[row]; j < rowPtr[row + 1]; ++j) {
            sum += values[j] * x[colIdx[j]];
        }
        b[row] -= sum;
    }
}

// Recursive block algorithm for SpTRSV
void spTRSVRecursiveBlock(const CSRMatrix& tri, float* d_b, float* d_x, int depth) {
    if (depth == 0) {
        // Base case: Perform standard sparse triangular solve
        int numThreads = 256;
        int numBlocks = (tri.rows + numThreads - 1) / numThreads;
        spTRSVKernel<<<numBlocks, numThreads>>> (tri.d_rowPtr, tri.d_colIdx, tri.d_values, d_b, d_x, tri.rows);
        hipDeviceSynchronize();
    }
    else {
        // Divide the matrix into top, bottom, and square blocks
        int mid = tri.rows / 2;

        // Extract submatrices
        CSRMatrix triTop = extractSubMatrix(tri, 0, mid, 0, mid);
        CSRMatrix rec = extractSubMatrix(tri, mid, tri.rows, 0, mid);
        CSRMatrix triBottom = extractSubMatrix(tri, mid, tri.rows, mid, tri.cols);

        // Allocate temporary vectors
        float* d_xTop, * d_bBottom;
        hipMalloc((void**)&d_xTop, mid * sizeof(float));
        hipMalloc((void**)&d_bBottom, (tri.rows - mid) * sizeof(float));
        hipMemset(d_xTop, 0, mid * sizeof(float));
        hipMemcpy(d_bBottom, d_b + mid, (tri.rows - mid) * sizeof(float), hipMemcpyDeviceToDevice);

        // Solve top triangular block
        spTRSVRecursiveBlock(triTop, d_b, d_xTop, depth - 1);

        // Update the right-hand side vector using the square block
        int numThreads = 256;
        int numBlocks = (rec.rows + numThreads - 1) / numThreads;
        spMVKernel << <numBlocks, numThreads >> > (rec.d_rowPtr, rec.d_colIdx, rec.d_values, d_xTop, d_bBottom, rec.rows);
        hipDeviceSynchronize();

        // Solve bottom triangular block
        float* d_xBottom;
        hipMalloc((void**)&d_xBottom, (tri.rows - mid) * sizeof(float));
        hipMemset(d_xBottom, 0, (tri.rows - mid) * sizeof(float));
        spTRSVRecursiveBlock(triBottom, d_bBottom, d_xBottom, depth - 1);

        // Combine the results
        hipMemcpy(d_x, d_xTop, mid * sizeof(float), hipMemcpyDeviceToDevice);
        hipMemcpy(d_x + mid, d_xBottom, (tri.rows - mid) * sizeof(float), hipMemcpyDeviceToDevice);

        // Free temporary vectors and submatrices
        hipFree(d_xTop);
        hipFree(d_bBottom);
        hipFree(d_xBottom);
        freeCSRMatrix(triTop);
        freeCSRMatrix(rec);
        freeCSRMatrix(triBottom);
    }
}

// Extract a submatrix from a CSR matrix
CSRMatrix extractSubMatrix(const CSRMatrix& mat, int startRow, int endRow, int startCol, int endCol) {
    CSRMatrix subMat;
    subMat.rows = endRow - startRow;
    subMat.cols = endCol - startCol;

    // Count non-zero elements in the submatrix
    int nnz = 0;
    for (int i = startRow; i < endRow; ++i) {
        for (int j = mat.d_rowPtr[i]; j < mat.d_rowPtr[i + 1]; ++j) {
            int col = mat.d_colIdx[j];
            if (col >= startCol && col < endCol) {
                nnz++;
            }
        }
    }
    subMat.nnz = nnz;

    // Allocate device memory for the submatrix
    allocateCSRMatrix(subMat, subMat.rows, subMat.cols, subMat.nnz);

    // Extract submatrix data
    std::vector<int> h_rowPtr(subMat.rows + 1, 0);
    std::vector<int> h_colIdx;
    std::vector<float> h_values;

    for (int i = startRow; i < endRow; ++i) {
        int newRow = i - startRow;
        h_rowPtr[newRow] = h_values.size();
        for (int j = mat.d_rowPtr[i]; j < mat.d_rowPtr[i + 1]; ++j) {
            int col = mat.d_colIdx[j];
            if (col >= startCol && col < endCol) {
                h_colIdx.push_back(col - startCol);
                h_values.push_back(mat.d_values[j]);
            }
        }
    }
    h_rowPtr[subMat.rows] = h_values.size();

    // Copy data to device
    hipMemcpy(subMat.d_rowPtr, h_rowPtr.data(), (subMat.rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(subMat.d_colIdx, h_colIdx.data(), h_colIdx.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(subMat.d_values, h_values.data(), h_values.size() * sizeof(float), hipMemcpyHostToDevice);

    return subMat;
}

// Allocate device memory for a CSR matrix
void allocateCSRMatrix(CSRMatrix& mat, int rows, int cols, int nnz) {
    mat.rows = rows;
    mat.cols = cols;
    mat.nnz = nnz;
    hipMalloc((void**)&mat.d_rowPtr, (rows + 1) * sizeof(int));
    hipMalloc((void**)&mat.d_colIdx, nnz * sizeof(int));
    hipMalloc((void**)&mat.d_values, nnz * sizeof(float));
}

// Free device memory for a CSR matrix
void freeCSRMatrix(CSRMatrix& mat) {
    hipFree(mat.d_rowPtr);
    hipFree(mat.d_colIdx);
    hipFree(mat.d_values);
}

// cuSPARSE implementation for SpTRSV
void cusparse_sptrsv(
    unsigned int n_rows,
    unsigned int nnz,
    const std::vector<unsigned int>& h_row_ptr,
    const std::vector<unsigned int>& h_col_ids,
    const std::vector<float>& h_data,
    const std::vector<float>& h_b,
    std::vector<float>& h_x,
    float& execution_time, float& throughput, float& bandwidth)
{
    // Allocate device memory
    unsigned int* d_row_ptr = nullptr;
    unsigned int* d_col_ids = nullptr;
    float* d_data = nullptr;
    float* d_b = nullptr;
    float* d_x = nullptr;

    hipMalloc(&d_row_ptr, h_row_ptr.size() * sizeof(unsigned int));
    hipMalloc(&d_col_ids, h_col_ids.size() * sizeof(unsigned int));
    hipMalloc(&d_data, h_data.size() * sizeof(float));
    hipMalloc(&d_b, h_b.size() * sizeof(float));
    hipMalloc(&d_x, n_rows * sizeof(float)); // Ensure size matches n_rows

    // Copy data from host to device
    hipMemcpy(d_row_ptr, h_row_ptr.data(), h_row_ptr.size() * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_ids, h_col_ids.data(), h_col_ids.size() * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_data, h_data.data(), h_data.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.data(), h_b.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_x, 0, n_rows * sizeof(float)); // Initialize solution vector to zero

    // cuSPARSE handle
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    // Matrix descriptor
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecB, vecX;
    hipsparseCreateCsr(&matA, n_rows, n_rows, nnz, d_row_ptr, d_col_ids, d_data,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    hipsparseCreateDnVec(&vecB, n_rows, d_b, HIP_R_32F);
    hipsparseCreateDnVec(&vecX, n_rows, d_x, HIP_R_32F);

    // SpSV descriptor
    hipsparseSpSVDescr_t spsvDescr;
    hipsparseSpSV_createDescr(&spsvDescr);

    // Buffer size and allocation
    size_t bufferSize;
    void* dBuffer = nullptr;
    float alpha = 1.0;
    hipsparseSpSV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecB, vecX,
        HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    // Analysis step
    hipsparseSpSV_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecB, vecX,
        HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr, dBuffer);

    // Timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int num_experiments = 5;
    hipEventRecord(start);
    for (int i = 0; i < num_experiments; ++i) {
        hipsparseSpSV_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecB, vecX,
            HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    milliseconds /= num_experiments; // Average time per kernel launch
    execution_time = milliseconds;

    // Copy result back to host
    hipMemcpy(h_x.data(), d_x, n_rows * sizeof(float), hipMemcpyDeviceToHost);

    // Calculate performance metrics
    throughput = 2.0f * nnz / (milliseconds / 1000.0f); // FLOPS
    float memory_bytes = (h_row_ptr.size() + h_col_ids.size() + h_data.size() + h_b.size() + h_x.size()) * sizeof(float);
    bandwidth = memory_bytes / (milliseconds / 1000.0f) / (1024.0f * 1024.0f * 1024.0f); // GB/s

    // Cleanup
    hipsparseSpSV_destroyDescr(spsvDescr);
    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnVec(vecB);
    hipsparseDestroyDnVec(vecX);
    hipsparseDestroy(handle);
    hipFree(dBuffer);
    hipFree(d_row_ptr);
    hipFree(d_col_ids);
    hipFree(d_data);
    hipFree(d_b);
    hipFree(d_x);
}

template <typename data_type>
bool compare_results(unsigned int y_size, const data_type* a, const data_type* b) {
    data_type numerator = 0.0;
    data_type denominator = 0.0;

    // Print all elements of a
    std::cerr << "Contents of array a:" << std::endl;
    for (unsigned int j = 0; j < 10; j++) {
        std::cerr << "a[" << j << "] = " << a[j] << std::endl;
    }

    // Print all elements of b
    std::cerr << "Contents of array b:" << std::endl;
    for (unsigned int j = 0; j < 10; j++) {
        std::cerr << "b[" << j << "] = " << b[j] << std::endl;
    }

    for (unsigned int i = 0; i < y_size; i++) {
        numerator += (a[i] - b[i]) * (a[i] - b[i]);
        denominator += b[i] * b[i];
    }

    const data_type error = numerator / denominator;

    if (error > 1e-3) {
        std::cerr << "ERROR: Mean Squared Error = " << error << std::endl;

        for (unsigned int i = 0; i < y_size; i++) {
            if (std::abs(a[i] - b[i]) > 1e-8) {
                std::cerr << "Mismatch at index " << i << ": a[" << i << "] = " << a[i] << ", b[" << i << "] = " << b[i] << std::endl;

                break;
            }
        }

        std::cerr.flush();
        return false;
    }

    std::cout << "Results match! Mean Squared Error = " << error << std::endl;
    return true;
}

// Function to read a matrix in Matrix Market format and convert it to CSR format
void read_matrix_market(const std::string& filename,
    std::vector<unsigned int>& row_ptr,
    std::vector<unsigned int>& col_ids,
    std::vector<float>& data,
    unsigned int& n_rows,
    unsigned int& n_cols) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        throw std::runtime_error("Could not open file: " + filename);
    }

    std::string line;
    bool is_symmetric = false;

    // Skip comments and check for symmetry flag
    while (std::getline(file, line)) {
        if (line[0] == '%') {
            // Check if the matrix is symmetric
            if (line.find("symmetric") != std::string::npos) {
                is_symmetric = true;
            }
            continue;
        }
        break;
    }

    // Read matrix dimensions and number of non-zero entries
    int rows, cols, nnz;
    std::istringstream iss(line);
    iss >> rows >> cols >> nnz;

    n_rows = rows;
    n_cols = cols;

    // Temporary storage for COO format
    std::vector<std::tuple<int, int, float>> coo_entries;
    coo_entries.reserve(nnz);

    int row, col;
    float value;
    while (file >> row >> col >> value) {
        // Convert 1-based indexing to 0-based indexing
        row -= 1;
        col -= 1;

        // For symmetric matrices, we need to consider both entries
        if (is_symmetric) {
            // If it's in the lower triangular part, add it directly
            if (row >= col) {
                coo_entries.push_back(std::make_tuple(row, col, value));
            }
            // If it's in the upper triangular part, add its symmetric counterpart
            else {
                coo_entries.push_back(std::make_tuple(col, row, value));
            }
        }
        // For non-symmetric matrices, only keep lower triangular entries
        else if (row >= col) {
            coo_entries.push_back(std::make_tuple(row, col, value));
        }
    }

    // Sort entries by row, then by column for CSR format
    std::sort(coo_entries.begin(), coo_entries.end());

    // Convert sorted COO to CSR format
    row_ptr.resize(rows + 1, 0);
    col_ids.clear();
    data.clear();

    col_ids.reserve(coo_entries.size());
    data.reserve(coo_entries.size());

    int current_row = -1;
    for (const auto& entry : coo_entries) {
        int r = std::get<0>(entry);
        int c = std::get<1>(entry);
        float v = std::get<2>(entry);

        // Fill in empty rows
        while (current_row < r) {
            current_row++;
            row_ptr[current_row] = col_ids.size();
        }

        col_ids.push_back(c);
        data.push_back(v);
    }

    // Fill in remaining row pointers
    while (current_row < rows) {
        current_row++;
        row_ptr[current_row] = col_ids.size();
    }
}


//
//int main() {
//    // File path to the MTX file
//    std::string filename = "matrices/chipcool0.mtx";
//
//    // Vectors to store CSR format data
//    std::vector<unsigned int> row_ptr, col_ids;
//    std::vector<float> data;
//    unsigned int n_rows, n_cols;
//
//    // Read the matrix from the file and convert it to CSR format
//    try {
//        read_matrix_market(filename, row_ptr, col_ids, data, n_rows, n_cols);
//    }
//    catch (const std::exception& e) {
//        std::cerr << e.what() << std::endl;
//        return EXIT_FAILURE;
//    }
//
//    // Right-hand side vector
//    std::vector<float> h_b(n_cols, 1.0f);
//    std::vector<float> h_x_manual(n_rows, 0.0f); // Result from manual implementation
//    std::vector<float> h_x_cusparse(n_rows, 0.0f); // Result from cuSPARSE implementation
//
//    //srand(static_cast<unsigned int>(time(NULL))); // Seed the random number generator
//    //for (unsigned int i = 0; i < n_cols; ++i) {
//    //    h_b[i] = static_cast<float>(rand()) / RAND_MAX; // Random value between 0 and 1
//    //}
//
//    // Performance metrics
//    float manual_time, manual_throughput, manual_bandwidth;
//    float cusparse_time, cusparse_throughput, cusparse_bandwidth;
//
//    //// Perform sparse triangular solve using manual implementation
//    //csr_sptrsv_host<float>(n_rows, col_ids, row_ptr, data, h_b, h_x_manual,
//    //    manual_time, manual_throughput, manual_bandwidth);
//
//    sptrsv_recursive_block_host(
//        n_rows, row_ptr, col_ids, data, h_b, h_x_manual,
//        manual_time, manual_throughput, manual_bandwidth
//    );
//
//    // Perform sparse triangular solve using cuSPARSE
//    cusparse_sptrsv(n_rows, data.size(), row_ptr, col_ids, data, h_b, h_x_cusparse,
//        cusparse_time, cusparse_throughput, cusparse_bandwidth);
//
//    // Compare results
//    bool results_match = compare_results(n_rows, h_x_manual.data(), h_x_cusparse.data());
//
//
//
//    if (results_match) {
//        std::cout << "Results match!" << std::endl;
//        std::cout << "Manual Implementation Performance:" << std::endl;
//        std::cout << "Execution time (ms): " << manual_time << std::endl;
//        std::cout << "Throughput (GFLOPS): " << manual_throughput / 1e9 << std::endl;
//        std::cout << "Memory bandwidth (GB/s): " << manual_bandwidth << std::endl;
//
//        std::cout << "cuSPARSE Implementation Performance:" << std::endl;
//        std::cout << "Execution time (ms): " << cusparse_time << std::endl;
//        std::cout << "Throughput (GFLOPS): " << cusparse_throughput / 1e9 << std::endl;
//        std::cout << "Memory bandwidth (GB/s): " << cusparse_bandwidth << std::endl;
//    }
//    else {
//        std::cout << "Results do not match!" << std::endl;
//    }
//
//    return 0;
//}

int main() {
    // Hardcoded 5x5 lower triangular matrix in CSR format
    unsigned int n_rows = 8;
    unsigned int n_cols = 8;

    std::vector<unsigned int> row_ptr = {
        0, 1, 3, 4, 6, 8, 11, 13, 15
    };

    std::vector<unsigned int> col_ids = {
        0,       // row 0
        0, 1,    // row 1
        2,       // row 2
        0, 3,    // row 3
        1, 4,  // row 4
        0, 3, 5, // row 5
        2, 6,    // row 6
        1, 7     // row 7
    };

    std::vector<float> data = {
        2.0,      // row 0
        -1.5, 3.0, // row 1
        4.0,      // row 2
        2.2, 1.0, // row 3
        5.5, 6.0, // row 4
        1.1, 3.3, 7.0, // row 5
        2.2, 8.0, // row 6
        4.4, 9.0  // row 7
    };

    // Right-hand side vector
    std::vector<float> h_b(n_cols, 1.0f);
    std::vector<float> h_x_manual(n_rows, 0.0f); // Result from manual implementation
    std::vector<float> h_x_cusparse(n_rows, 0.0f); // Result from cuSPARSE implementation

    // Performance metrics
    float manual_time, manual_throughput, manual_bandwidth;
    float cusparse_time, cusparse_throughput, cusparse_bandwidth;

    sptrsv_recursive_block_host(
        n_rows, row_ptr, col_ids, data, h_b, h_x_manual,
        manual_time, manual_throughput, manual_bandwidth
    );

    // Perform sparse triangular solve using cuSPARSE
    cusparse_sptrsv(n_rows, data.size(), row_ptr, col_ids, data, h_b, h_x_cusparse,
        cusparse_time, cusparse_throughput, cusparse_bandwidth);

    // Compare results
    bool results_match = compare_results(n_rows, h_x_manual.data(), h_x_cusparse.data());

    // Print the matrix in CSR format
    std::cout << "Matrix in CSR format:" << std::endl;
    for (unsigned int i = 0; i < n_rows; ++i) {
        for (unsigned int j = row_ptr[i]; j < row_ptr[i + 1]; ++j) {
            std::cout << "Row " << i << ", Col " << col_ids[j]
                << ", Value " << data[j] << std::endl;
        }
    }

    if (results_match) {
        std::cout << "Results match!" << std::endl;
        std::cout << "Manual Implementation Performance:" << std::endl;
        std::cout << "Execution time (ms): " << manual_time << std::endl;
        std::cout << "Throughput (GFLOPS): " << manual_throughput / 1e9 << std::endl;
        std::cout << "Memory bandwidth (GB/s): " << manual_bandwidth << std::endl;

        std::cout << "cuSPARSE Implementation Performance:" << std::endl;
        std::cout << "Execution time (ms): " << cusparse_time << std::endl;
        std::cout << "Throughput (GFLOPS): " << cusparse_throughput / 1e9 << std::endl;
        std::cout << "Memory bandwidth (GB/s): " << cusparse_bandwidth << std::endl;
    }
    else {
        std::cout << "Results do not match!" << std::endl;
    }

    return 0;
}
